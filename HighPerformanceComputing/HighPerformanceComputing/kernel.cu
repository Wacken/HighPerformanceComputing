#include "hip/hip_runtime.h"
﻿__host__ void matrixMult(float *A, float *B, float *C, int n) 
{
	int size = n * n * sizeof(float);
	float* Ad; float* Bd; float* Cd;
	hipMalloc((void**)&Ad, size);
	hipMalloc((void**)&Bd, size);
	hipMalloc((void**)&Cd, size);
	hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
	hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);
	hipMemcpy(Cd, C, size, hipMemcpyHostToDevice);
	/* ... perform multiplication on device ... */
	hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);
	hipFree(Ad); hipFree(Bd); hipFree(Cd);
}

__global__ void matrixMultKernel(float* Ad, float* Bd, float* Cd, int n) 
{
	int i = threadIdx.x;
	int k = threadIdx.y;
	float Celem = 0;
	for (int j = 0; j<n; j++) {
		float Aelem = Ad[i*n + j];
		float Belem = Bd[j*n + k];
		Celem += Aelem * Belem;
	}
	Cd[i*n + k] += Celem;
}

int main() 
{
	dim3 dimBlock(TILE_SIZE, TILE_SIZE);
	dim3 dimGrid(n / TILE_SIZE, n / TILE_SIZE);
	matrixMultKernel << <dimGrid, dimBlock >> >(Ad, Bd, Cd, n);

}