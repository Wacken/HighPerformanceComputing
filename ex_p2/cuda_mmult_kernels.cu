#include "hip/hip_runtime.h"
#include "cuda_mmult_kernels.h"

/* 
 * matrix multiplication C += A*B 
 *  -> CUDA kernel
 *     (implementation adopted from Kirk&Hwu: 
 *      "Programming Massively Parallel Processors, chapter 4)
 *  -> Features: none (basic tiled version, using only global memory)
 */
__global__ void matrixMultKernel_global(float* Ad, float* Bd, float* Cd, int n)
{
   int i = threadIdx.x;
   int k = threadIdx.y;
   
   float Celem = 0;
   
   for(int j=0; j<n; j++) {
      float Aelem = Ad[i*n+j];
      float Belem = Bd[j*n+k];
      Celem += Aelem*Belem;
   }
   
   Cd[i*n+k] += Celem;
}

/* 
 * matrix multiplication C += A*B 
 *  -> CUDA kernel
 *     (implementation adopted from Kirk&Hwu: 
 *      "Programming Massively Parallel Processors, chapter 5)
 *  -> Features:
 *     - tiled matrix multiplication with use of shared memory
 */
__global__ void matrixMultKernel_tiled(float* Ad, float* Bd, float* Cd, int n)
{
      /* TODO: implement tiled matrix multiplication */

}
